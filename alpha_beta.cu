#include "hip/hip_runtime.h"
#include "alpha_beta.h"

__device__ void cudaSearch(Node *node, int player, int maximizer, int ply) {
    if (ply == 0){
        node->alpha = diffeval(maximizer, node->board);
        node->beta = diffeval(maximizer, node->board);
        return;
    }

    int * moves = legalmoves(node->player, node->board);
    if(moves[0] == 0) return;

    for (int i = 1; i < moves[0]; i++) {
        int * newboard = copyboard(node->board);
        int move = moves[i];
        makemove(moves[1], opponent(node->player), newboard);
        int ntm = nexttoplay(newboard, opponent(node->player), 0);
        if (ntm == 0){
            node->alpha = diffeval(node->player, node->board);
            node->beta = diffeval(node->player, node->board);
            return;
        }
        ntm = cudanexttoplay(newboard, opponent(node->player), 0);
    
        // makemove(move, node->player, newboard);
    
        Node *newNode = node;
        newNode->move = move;
        newNode->player = ntm;
        newNode->alpha = node->alpha;
        newNode->beta = node->beta;
        newNode->board = newboard;
        newNode->parent = node;

        // search child
        cudaSearch(newNode, ntm, maximizer, ply - 1);

        if (player == ntm) {
            node->beta = min(node->beta, newNode->alpha);
        } 
        if (opponent(player) == ntm){
            node->alpha = max(node->alpha, newNode->beta);
        }

        if (node->alpha >= node->beta) {
            return;
        }
        free(newNode);
    }
}

__global__
void cudaTreeKernel(int * moves, int * board, int * values, int player, int maximizer,
    int alpha, int beta, int ply) {
    // only one thread does high-level tasks
    if (threadIdx.x == 0) {
        // make one new node per block
        if(moves[0] == 0) return;
        int move = moves[blockIdx.x];
        int * newboard = copyboard(board);
        makemove(move, player, newboard);
        int ntm = cudanexttoplay(newboard, player, 0);
        
        Node *newNode = new Node;
        newNode->move = move;
        newNode->player = ntm;
        newNode->alpha = alpha;
        newNode->beta = beta;
        newNode->board = newboard;

        cudaSearch(newNode, player, maximizer, ply);

        // update the values we care about - if the parent node is a maximizing node, 
        // it cares about the child alpha values
        if (player == maximizer) {
            values[blockIdx.x] = newNode->beta;
        } 
        if (opponent(player) == maximizer){
            values[blockIdx.x] = newNode->alpha;
        }
        free(newNode);
    }
}

void cudaMinMaxKernel(int * moves, int * board, int *values, int player, int maximizer, int alpha, int beta, int numMoves, int ply) {

    cudaTreeKernel<<<numMoves, 32>>>(moves, board, values, player, maximizer, alpha, beta, ply);
}

int search(Node *node, int maximizer, int ply) {
    // Do not search any deeper
    if (ply == 0){
        node->alpha = diffeval(maximizer, node->board);
        node->beta = diffeval(maximizer, node->board);
        return NULL;
    }

    // make copy of board and find moves
    int * newboard = copyboard(node->board);
    int * moves = legalmoves(node->player, node->board);
    makemove(moves[1], opponent(node->player), newboard);
    int ntm = cudanexttoplay(newboard, node->player, 0);
    
    Node *newNode = node;
    newNode->move = moves[1];
    newNode->player = ntm;
    newNode->alpha = node->alpha;
    newNode->beta = node->beta;
    newNode->board = newboard;
    newNode->parent = node;

    int best = search(newNode,maximizer, ply - 1);

    int *values;

    values = (int *)calloc(moves[0], sizeof(int));

    if (node->player == maximizer) {
        values[0] = newNode->alpha;
    } 
    if (opponent(node->player) == maximizer) {
        values[0] = newNode->beta;
    }

    /* GPU search the rest of the child nodes */
    int numMoves = moves[0];
    int *dev_moves;
    int *dev_board;
    int *dev_values;
    int *tmoves = (int *)malloc(numMoves * sizeof(int));
    for (int i = 1; i < moves[0]; i++) {
        tmoves[i] = moves[i];
    }

    hipMalloc((void **) &dev_moves, numMoves * sizeof(int));
    hipMalloc((void **) &dev_board, BOARDSIZE * sizeof(int));
    hipMalloc((void **) &dev_values, numMoves * sizeof(int));

    hipMemcpy(dev_board, &(node->board), BOARDSIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_moves, tmoves, numMoves * sizeof(int), hipMemcpyHostToDevice);

    hipMemset(dev_values, 0, (numMoves) * sizeof(int));

    // call kernel to search the rest of the children in parallel
    cudaMinMaxKernel(dev_moves, dev_board, dev_values, ntm, maximizer, 
        node->alpha, node->beta, numMoves, ply);

    // copy remaining child values into host array
    hipMemcpy(values, dev_values, numMoves * sizeof(int), hipMemcpyDeviceToHost);

    // find the best move
    int index = 1;
    if (node->player == maximizer) {
        int best =  WIN+1;
        for (int i = 1; i <= numMoves; i++) {
            if (values[i] < best) {
                best = values[i];
                index = i;
            }
        }
        node->beta = best;
    } else {
        int best = LOSS - 1;
        for (int i = 1; i <= numMoves; i++) {
            if (values[i] > best) {
                best = values[i];
                index = i;
            }
        }
        node->alpha = best;
    }
    // printf("%d\n", moves[index]);

    hipFree(dev_values);
    hipFree(dev_board);
    hipFree(dev_moves);

    return moves[index];
}
    