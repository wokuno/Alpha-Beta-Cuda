
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10
#define BLOCKSIZE 10

void minmaxCuda(double *max, double *min, double *a, float &time);

__global__ void minmaxKernel(double *max, double *min, double *a) {
	__shared__ double maxtile[BLOCKSIZE];
	__shared__ double mintile[BLOCKSIZE];
	
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	maxtile[tid] = a[i];
	mintile[tid] = a[i];
	__syncthreads();
	
	// strided index and non-divergent branch
	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		int index = 2 * s * tid;
		if (index < blockDim.x) {
			if (maxtile[tid + s] > maxtile[tid])
				maxtile[tid] = maxtile[tid + s];
			if (mintile[tid + s] < mintile[tid])
				mintile[tid] = mintile[tid + s];
		}
		__syncthreads();
	}
	
	if (tid == 0) {
		max[blockIdx.x] = maxtile[0];
		min[blockIdx.x] = mintile[0];
	}
}

__global__ void finalminmaxKernel(double *max, double *min) {
	__shared__ double maxtile[BLOCKSIZE];
	__shared__ double mintile[BLOCKSIZE];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	maxtile[tid] = max[i];
	mintile[tid] = min[i];
	__syncthreads();
	
	// strided index and non-divergent branch
	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		int index = 2 * s * tid;
		if (index < blockDim.x) {
			if (maxtile[tid + s] > maxtile[tid])
				maxtile[tid] = maxtile[tid + s];
			if (mintile[tid + s] < mintile[tid])
				mintile[tid] = mintile[tid + s];
		}
		__syncthreads();
	}
	
	if (tid == 0) {
		max[blockIdx.x] = maxtile[0];
		min[blockIdx.x] = mintile[0];
	}
}

void minmaxCuda(double *max, double *min, double *a, float &time)
{

    double *dev_a = 0;
    double *dev_max = 0;
	double *dev_min = 0;
	float milliseconds = 0;

	dim3 dimBlock(BLOCKSIZE);
    dim3 dimGrid(N);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipMalloc((void**)&dev_max, N * sizeof(double));


	hipMalloc((void**)&dev_min, N * sizeof(double));
   
    hipMalloc((void**)&dev_a, N * N * sizeof(double));

    hipMemcpy(dev_a, a, N * N * sizeof(double), hipMemcpyHostToDevice);


	hipEventRecord(start);
    minmaxKernel<<<dimGrid, dimBlock>>>(dev_max, dev_min, dev_a);
	finalminmaxKernel<<<1, dimBlock>>>(dev_max, dev_min);
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	hipDeviceSynchronize();

    hipMemcpy(max, dev_max, N * sizeof(double), hipMemcpyDeviceToHost);


	hipMemcpy(min, dev_min, N * sizeof(double), hipMemcpyDeviceToHost);

	hipEventElapsedTime(&milliseconds, start, stop);
	time = milliseconds;

    hipFree(dev_max);
	hipFree(dev_min);
    hipFree(dev_a);
}

